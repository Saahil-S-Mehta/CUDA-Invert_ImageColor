#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Inversion_CUDA.h"

__global__ void Inversion_CUDA(unsigned char* Image, int Channels);
__global__ void Gray_CUDA(unsigned char* Image, int Channels);

void Image_Inversion_CUDA(unsigned char* Input_Image, int Height, int Width, int Channels) {
	unsigned char* Dev_Input_Image = NULL;

	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	Inversion_CUDA << <Grid_Image, 8 >> > (Dev_Input_Image, Channels);

	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	hipFree(Dev_Input_Image);
}

void Image_Gray_CUDA(unsigned char* Input_Image, int Height, int Width, int Channels) {
	unsigned char* Dev_Input_Image = NULL;

	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	Gray_CUDA << <Grid_Image, 1 >> > (Dev_Input_Image, Channels);

	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	hipFree(Dev_Input_Image);
}

__global__ void Gray_CUDA(unsigned char* Image, int Channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * Channels;

	unsigned char temp;

	for (int i = 0; i < Channels; i++) {
		temp += Image[idx + i];
	}

	//temp = temp / 3;

	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = (unsigned char)temp/3;
	}
}

__global__ void Inversion_CUDA(unsigned char* Image, int Channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
	}
}
